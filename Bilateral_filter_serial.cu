#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
//#include <hip/hip_runtime.h>
//#include <>
//#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <helper_image.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include <opencv2\opencv.hpp>
#include <opencv2\highgui\highgui.hpp>
#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui_c.h>

using namespace cv;
using namespace std;
/*
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void filter(uchar *d_data){
	int t = blockIdx.x*blockDim.x + threadIdx.x;
	d_data[t] = 100;
}
*/
int main(int argc, char** argv){
	const Mat img = cv::imread("peppers.jpg",CV_LOAD_IMAGE_UNCHANGED);
	Mat img2 = img.clone();
	Mat img3 = img.clone();

	if(img.empty()){
		cout<<"Error: Image not supported"<<endl;
		return -1;
	}

	else{
		uchar *input = img.data;
		const int rows =(const int)img.rows;
		const int cols = (const int)img.cols;
		const int step = img.step;
		int channels = img.channels();
		std::cout<< cols <<" x "<<rows<<"Step Size: "<<step<<endl;
		if(img.depth()==CV_8U) cout << "Unsigned char image" << endl;
		std::cout<<"Number of channels: "<<img.channels()<<endl;
		std::cout<<"Is data continuous: "<<img.isContinuous()<<endl;
		//bilateralFilter(img, img2, 7, 10, 10);

/*
		uchar * d_data;
		unsigned int size = channels*rows*cols*sizeof(uchar);
		
		hipMalloc((void**)&d_data, size);
		hipMemcpy( d_data, img.data, size, hipMemcpyHostToDevice);
	
		float time;
		hipEvent_t start, stop;

		HANDLE_ERROR( hipEventCreate(&start) );
		HANDLE_ERROR( hipEventCreate(&stop) );
		HANDLE_ERROR( hipEventRecord(start, 0) );

		//filter<<<rows, channels*cols>>>(d_data);
		
		hipDeviceSynchronize();
		
		HANDLE_ERROR( hipEventRecord(stop, 0) );
		HANDLE_ERROR( hipEventSynchronize(stop) );
		HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
		printf("Time to generate:  %f ms \n", time);
		//hipMemcpy(img.data, d_data, size, hipMemcpyDeviceToHost);
		
*/
		const unsigned int kr = 7;
		const int ks = 2*kr+1;
		double sigs=9;
		double sigr=9;
		double g[ks][ks],d[ks][ks],gdist[ks][ks];
		uchar I[ks][ks], a[ks][ks];
		uchar *input2 = img2.data;
		//unsigned char *input4 = (unsigned char*)(img4.data);
		double norm_gdist=0;
		for(int i=0; i<ks; i++){
			for(int j=0; j<ks; j++){
				d[i][j] = (kr-i)*(kr-i)+(kr-j)*(kr-j);
				//cout<<d[i][j]<<"   ";
			}
			//cout<<endl;
		}
		
		for(int i=0; i<ks;i++){
			for(int j=0;j<ks;j++){
				gdist[i][j]=(1/(sigs*sqrt(2*3.142)))*exp(-d[i][j]/(2*sigs*sigs));
				norm_gdist += gdist[i][j];
				//cout<<gdist[i][j]<<"   ";
			}
			//cout<<endl;
		}
			
			int l=rows*step;
/*		
			for(int y=0; y<rows; y++)
				for(int x=0; x<cols; x++){
					img_2D[y][x]=input[y*step+channels*x];
					img_2D[y][x+1]=input[y*step+channels*x+1];
					img_2D[y][x+2]=input[y*step+channels*x+2];
				}
*/				

		for(int i=0; i<l; i++){
			int x=(i%step);
			int y=(i/step);
			for(int p=0; p<kr; p++){
				for(int q=0; q<kr; q++){
					if(x-channels*q<0 || y-p<0 || x+channels*q>=l || y+p>=l){
						I[p][q]=input[i];
						a[p][q]=0;
						}

						else{
							a[kr+p][kr+q]=input[i+p*step+channels*q];
							I[kr+p][kr+q]=a[kr+p][kr+q]-a[kr][kr];

							a[kr-p][kr+q]=input[i-p*step+channels*q];
							I[kr-p][kr+q]=a[kr-p][kr+q]-a[kr][kr];

							a[kr+p][kr-q]=input[i+p*step-channels*q];
							I[kr+p][kr-q]=a[kr+p][kr-q]-a[kr][kr];

							a[kr-p][kr-q]=input[i-p*step-channels*q];
							I[kr-p][kr-q]=a[kr-p][kr-q]-a[kr][kr];
							
						}
			//			std::cout<<(int)I[p][q]<<"  ";
				}
			//	std::cout<<endl;
			}
					/*					else{
						a[kr-p][kr-q]= input[i-p*step-channels*q];
						I[kr-p][kr-q]= input[i-p*step-channels*q]-input[i];
					//	cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}

					if(i-channels*q<0){
						I[kr][kr+q]=0;
						a[kr][kr+q]=0;
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}
					else{
						a[kr-p][kr+q]=input[i-p*step+channels*q];
						I[kr-p][kr+q]=input[i-p*step+channels*q]-input[i];
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}

					if((i+p*step-channels*q<0)||(i+p*step-channels*q>=step)){
						I[kr+p][kr-q]=0;
						a[kr+p][kr-q]=0;
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}
					else{
						a[kr+p][kr-q]=input[i+p*step-channels*q];
						I[kr+p][kr-q]=input[i+p*step-channels*q]-input[i];
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}

					if((i+p*step+channels*q<0)||(i+p*step+channels*q>=step)){
						I[kr+p][kr+q]=0;
						a[kr+p][kr+q]=0;
						//cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}
					else{
						a[kr+p][kr+q]=input[i+p*step+channels*q];
						I[kr+p][kr+q]=input[i+p*step+channels*q]-input[i];
					}
					
				}
			}
			
			//k=i%step;
			//j=(k-i*step)%channels;
*/			
			
		
			double sum=0;
			for(int t=0; t<ks;t++)
				for(int j=0;j<ks;j++){
				//cout<<endl<<g[t][j]<<"initial"<<endl;
				g[t][j]=(1/(sigr*sqrt(2*3.142)))*exp(-((I[t][j]*I[t][j]))/(2*sigr*sigr));
				
				//cout<<endl<<g[t][j]<<"final"<<endl;				
			}
				//cout<<endl<<sum<<"final"<<endl;	

			double answer=0;
			for(int q=0;q<ks;q++)
				for(int w=0;w<ks;w++){
					answer+=a[q][w]*gdist[q][w]*g[q][w];
					sum+=g[q][w]*gdist[q][w];
					//cout<<answer/sum<<"ansmwer"<<endl;
				}

			input2[i]= (uchar)(answer/sum);
		}

		///////////////////////////////////////////////////////
/*		for (int y=0; y<rows; y++){
			for(int x=0; x<channels*cols; x++){

				for(int p=0; p<kr; p++){
					for(int q=0; q<kr; q++){
						if(x-channels*q<0 || y-p<0 || x+channels*q>=l || y+p>=l){
							I[p][q]=img_2D[y][x];
							a[p][q]=0;
						}

						else{
							a[kr+p][kr+q]=img_2D[y+p][x+channels*q];
							I[kr+p][kr+q]=a[kr+p][kr+q]-a[kr][kr];

							a[kr-p][kr+q]=img_2D[y-p][x+channels*q];
							I[kr-p][kr+q]=a[kr-p][kr+q]-a[kr][kr];

							a[kr+p][kr-q]=img_2D[y+p][x-channels*q];
							I[kr+p][kr-q]=a[kr+p][kr-q]-a[kr][kr];

							a[kr-p][kr-q]=img_2D[y-p][x-channels*q];
							I[kr-p][kr-q]=a[kr-p][kr-q]-a[kr][kr];
							
						}
					}
						double norm_g=0, answer=0;
						for(int i=0; i<ks;i++){
							for(int j=0;j<ks;j++){
								g[i][j]=gdist[i][j]*exp(-I[i][j]*I[i][j]/(2*sigr*sigr));
								answer += a[i][j]*g[i][j];
								norm_g += g[i][j];
								//cout<<gdist[i][j]<<"   ";
							}
							//cout<<endl;
						}
						uchar total = (uchar)(answer/norm_g);
						img_2D[y][x] = total;
					}
				}
		}

		for(int y=0; y<rows; y++)
				for(int x=0; x<cols; x++){
					input[y*step+channels*x]=img_2D[y][x];
					input[y*step+channels*x+1]=img_2D[y][x+1];
					input[y*step+channels*x+2]=img_2D[y][x+2];
				}
*/
		cv::namedWindow("Original",CV_WINDOW_AUTOSIZE);
		cv::imshow("Original", img);
		cv::namedWindow("Sampled",CV_WINDOW_AUTOSIZE);
		cv::imshow("Sampled", img2);
		cv::waitKey(0);
		cv::destroyAllWindows();		

	}
}