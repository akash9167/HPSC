#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <helper_image.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>
#include <opencv2\opencv.hpp>
#include <opencv2\highgui\highgui.hpp>

using namespace cv;
using namespace std;
/*
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void filter(uchar *d_data){
	int t = blockIdx.x*blockDim.x + threadIdx.x;
	d_data[t] = 100;
}
*/
int main(int argc, char** argv){
	const Mat img = cv::imread("scar1.jpg",CV_LOAD_IMAGE_UNCHANGED);
	Mat img2 = img.clone();
	Mat img3 = img.clone();

	if(img.empty()){
		cout<<"Error: Image not supported"<<endl;
		return -1;
	}

	else{
		uchar *input = img.data;

		int rows = img.rows,
			cols = img.cols,
			step = img.step,
			rows_d, cols_d, step_d;

		int channels = img.channels();
		std::cout<< cols <<" x "<<rows<<"Step Size: "<<step<<endl;
		if(img.depth()==CV_8U) cout << "Unsigned char image" << endl;
		std::cout<<"Number of channels: "<<img.channels()<<endl;
		std::cout<<"Is data continuous: "<<img.isContinuous()<<endl;
		size_t total_size = rows*cols*3*sizeof(uchar);
		
		uchar *input_d, *output_d;
		
		const unsigned int kr = 8;
		const int ks = 2*kr+1;
		double sigs=20;
		double sigr=20;
		double g[ks][ks],d[ks][ks],gdist[ks][ks];
		uchar I[ks][ks], a[ks][ks];
		uchar *input2 = img2.data;
		
		double norm_gdist=0;

		for(int i=0; i<ks; i++){
			for(int j=0; j<ks; j++){
				d[i][j] = (kr-i)*(kr-i)+(kr-j)*(kr-j);
				float den = 1/(sigs*sqrt(2*3.142));
				float exp_=-d[i][j]/(2*sigs*sigs);
				gdist[i][j]=den*exp(exp_);
				norm_gdist += gdist[i][j];
				//cout<<d[i][j]<<"   ";
			}
			//cout<<endl;
		}
/*
		hipMemcpy(input_d, input, total_size,hipMemcpyHostToDevice);
		hipMemcpy(&rows_d, &rows, sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(&cols_d, &cols, sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(&step_d, &step, sizeof(int),hipMemcpyHostToDevice);
		
		hipMalloc((void**)input_d,	total_size);
		hipMalloc((void**)rows_d,	sizeof(int));
		hipMalloc((void**)cols_d,	sizeof(int));
		hipMalloc((void**)step_d,	sizeof(int));
		hipMalloc((void**)output_d,total_size);
		hipMalloc((void**)output_d,total_size);
*/
/*
		uchar * d_data;
		unsigned int size = channels*rows*cols*sizeof(uchar);
		
		hipMalloc((void**)&d_data, size);
		hipMemcpy( d_data, img.data, size, hipMemcpyHostToDevice);
	
		float time;
		hipEvent_t start, stop;

		HANDLE_ERROR( hipEventCreate(&start) );
		HANDLE_ERROR( hipEventCreate(&stop) );
		HANDLE_ERROR( hipEventRecord(start, 0) );

		//filter<<<rows, channels*cols>>>(d_data);
		
		hipDeviceSynchronize();
		
		HANDLE_ERROR( hipEventRecord(stop, 0) );
		HANDLE_ERROR( hipEventSynchronize(stop) );
		HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
		printf("Time to generate:  %f ms \n", time);
		//hipMemcpy(img.data, d_data, size, hipMemcpyDeviceToHost);
		
*/
			
			int l=rows*step;
/*		
			for(int y=0; y<rows; y++)
				for(int x=0; x<cols; x++){
					img_2D[y][x]=input[y*step+channels*x];
					img_2D[y][x+1]=input[y*step+channels*x+1];
					img_2D[y][x+2]=input[y*step+channels*x+2];
				}
*/				

		for(int i=0; i<l; i++){
			int x=(i%step);
			int y=(i/step);
			for(int p=0; p<kr; p++){
				for(int q=0; q<kr; q++){
					if(x-channels*q<0 || y-p<0 || x+channels*q>=l || y+p>=l || i+p*step+channels*q>=l ||i+p*step-channels*q>=l||i-p*step+channels*q>=l){
						I[kr+p][kr+q]=input[i];
						a[kr+p][kr+q]=0;

						I[kr-p][kr+q]=input[i];
						a[kr-p][kr+q]=0;

						I[kr+p][kr-q]=input[i];
						a[kr+p][kr-q]=0;

						I[kr-p][kr-q]=input[i];
						a[kr-p][kr-q]=0;
						}

						else{
							a[kr+p][kr+q]=input[i+p*step+channels*q];
							I[kr+p][kr+q]=a[kr+p][kr+q]-a[kr][kr];

							a[kr-p][kr+q]=input[i-p*step+channels*q];
							I[kr-p][kr+q]=a[kr-p][kr+q]-a[kr][kr];

							a[kr+p][kr-q]=input[i+p*step-channels*q];
							I[kr+p][kr-q]=a[kr+p][kr-q]-a[kr][kr];

							a[kr-p][kr-q]=input[i-p*step-channels*q];
							I[kr-p][kr-q]=a[kr-p][kr-q]-a[kr][kr];
							
						}
			//			std::cout<<(int)I[p][q]<<"  ";
				}
			//	std::cout<<endl;
			}
					/*					else{
						a[kr-p][kr-q]= input[i-p*step-channels*q];
						I[kr-p][kr-q]= input[i-p*step-channels*q]-input[i];
					//	cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}

					if(i-channels*q<0){
						I[kr][kr+q]=0;
						a[kr][kr+q]=0;
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}
					else{
						a[kr-p][kr+q]=input[i-p*step+channels*q];
						I[kr-p][kr+q]=input[i-p*step+channels*q]-input[i];
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}

					if((i+p*step-channels*q<0)||(i+p*step-channels*q>=step)){
						I[kr+p][kr-q]=0;
						a[kr+p][kr-q]=0;
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}
					else{
						a[kr+p][kr-q]=input[i+p*step-channels*q];
						I[kr+p][kr-q]=input[i+p*step-channels*q]-input[i];
//						cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}

					if((i+p*step+channels*q<0)||(i+p*step+channels*q>=step)){
						I[kr+p][kr+q]=0;
						a[kr+p][kr+q]=0;
						//cout<<"I[][]: "<<(int)I[kr+p][kr+q]<<endl;
					}
					else{
						a[kr+p][kr+q]=input[i+p*step+channels*q];
						I[kr+p][kr+q]=input[i+p*step+channels*q]-input[i];
					}
					
				}
			}
			
			//k=i%step;
			//j=(k-i*step)%channels;
*/			
			
		
			double sum=0;
			for(int t=0; t<ks;t++)
				for(int j=0;j<ks;j++){
				//cout<<endl<<g[t][j]<<"initial"<<endl;
				g[t][j]=(1/(sigr*sqrt(2*3.142)))*exp(-((I[t][j]*I[t][j]))/(2*sigr*sigr));
				
				//cout<<endl<<g[t][j]<<"final"<<endl;				
			}
				//cout<<endl<<sum<<"final"<<endl;	

			double answer=0;
			for(int q=0;q<ks;q++)
				for(int w=0;w<ks;w++){
					answer+=a[q][w]*gdist[q][w]*g[q][w];
					sum+=g[q][w]*gdist[q][w];
					//cout<<answer/sum<<"ansmwer"<<endl;
				}

			input2[i]= (uchar)(answer/sum);
		}

		///////////////////////////////////////////////////////
/*		for (int y=0; y<rows; y++){
			for(int x=0; x<channels*cols; x++){

				for(int p=0; p<kr; p++){
					for(int q=0; q<kr; q++){
						if(x-channels*q<0 || y-p<0 || x+channels*q>=l || y+p>=l){
							I[p][q]=img_2D[y][x];
							a[p][q]=0;
						}

						else{
							a[kr+p][kr+q]=img_2D[y+p][x+channels*q];
							I[kr+p][kr+q]=a[kr+p][kr+q]-a[kr][kr];

							a[kr-p][kr+q]=img_2D[y-p][x+channels*q];
							I[kr-p][kr+q]=a[kr-p][kr+q]-a[kr][kr];

							a[kr+p][kr-q]=img_2D[y+p][x-channels*q];
							I[kr+p][kr-q]=a[kr+p][kr-q]-a[kr][kr];

							a[kr-p][kr-q]=img_2D[y-p][x-channels*q];
							I[kr-p][kr-q]=a[kr-p][kr-q]-a[kr][kr];
							
						}
					}
						double norm_g=0, answer=0;
						for(int i=0; i<ks;i++){
							for(int j=0;j<ks;j++){
								g[i][j]=gdist[i][j]*exp(-I[i][j]*I[i][j]/(2*sigr*sigr));
								answer += a[i][j]*g[i][j];
								norm_g += g[i][j];
								//cout<<gdist[i][j]<<"   ";
							}
							//cout<<endl;
						}
						uchar total = (uchar)(answer/norm_g);
						img_2D[y][x] = total;
					}
				}
		}

		for(int y=0; y<rows; y++)
				for(int x=0; x<cols; x++){
					input[y*step+channels*x]=img_2D[y][x];
					input[y*step+channels*x+1]=img_2D[y][x+1];
					input[y*step+channels*x+2]=img_2D[y][x+2];
				}
*/
		cv::namedWindow("Original",CV_WINDOW_AUTOSIZE);
		cv::imshow("Original", img);
		cv::namedWindow("Sampled",CV_WINDOW_AUTOSIZE);
		cv::imshow("Sampled", img2);
		cv::waitKey(0);
		cv::destroyAllWindows();		

	}
}
